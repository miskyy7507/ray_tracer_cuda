#include "hip/hip_runtime.h"
#include "Sphere.cuh"

__device__ Sphere::Sphere(const Vector3 &_center, float _radius) : center(_center), radius(_radius) {}

__device__ bool Sphere::hit(const Ray &r, float t_min, float t_max, HitRecord &rec) const {
    Vector3 oc = center - r.origin(); // co to jest?

    float a = r.direction().dot(r.direction());
    // float b = -2.0f * ray.direction().dot(oc);
    float h = r.direction().dot(oc);
    float c = oc.dot(oc) - radius * radius;

    // float discriminant = b*b - 4*a*c;
    float discriminant = h*h - a*c;

    if (discriminant < 0.0f) {
        return false;
    }

    float sqrtd = sqrtf(discriminant);

    float root;
    root = (h - sqrtd) / a;
    if (root <= t_min || root >= t_max) {
        root = (h + sqrtd) / a;
        if (root <= t_min || root >= t_max) {
            return false;
        }
    }

    // float root1 = (h - sqrtd) / a;
    // float root2 = (h + sqrtd) / a;
    //
    // bool root1_in_t = root1 > t_min && root1 < t_max;
    // bool root2_in_t = root2 > t_min && root2 < t_max;
    //
    // return !(root1_in_t || root2_in_t);

    rec.point = r.point_at(root);
    rec.normal = (rec.point - center) / radius;
    rec.t = root;

    return true;
}
