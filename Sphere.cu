#include "hip/hip_runtime.h"
#include "Sphere.cuh"

__device__ Sphere::Sphere(const Vector3 &_center, float _radius, Material* _material)
    : center(_center), radius(_radius), material(_material) {}

__device__ bool Sphere::hit(const Ray &r, float t_min, float t_max, HitRecord &rec) const {
    Vector3 oc = this->center - r.origin(); // co to jest?

    float a = r.direction().dot(r.direction());
    // float b = -2.0f * ray.direction().dot(oc);
    float h = r.direction().dot(oc);
    float c = oc.dot(oc) - this->radius * this->radius;

    // float discriminant = b*b - 4*a*c;
    float discriminant = h*h - a*c;

    if (discriminant < 0.0f) {
        return false;
    }

    float sqrtd = sqrtf(discriminant);

    float root;
    root = (h - sqrtd) / a;
    if (root <= t_min || root >= t_max) {
        root = (h + sqrtd) / a;
        if (root <= t_min || root >= t_max) {
            return false;
        }
    }

    // float root1 = (h - sqrtd) / a;
    // float root2 = (h + sqrtd) / a;
    //
    // bool root1_in_t = root1 > t_min && root1 < t_max;
    // bool root2_in_t = root2 > t_min && root2 < t_max;
    //
    // return !(root1_in_t || root2_in_t);

    rec.point = r.point_at(root);
    rec.normal = (rec.point - this->center) / this->radius;
    rec.material = this->material;
    rec.t = root;

    return true;
}

__global__ void create_sphere(Vector3 center, float radius, int mat_index, Material** mat, Hittable** list, size_t index) {
    list[index] = new Sphere(center, radius, mat[mat_index]);
}