#include "hip/hip_runtime.h"
#include "Matte.cuh"

__device__ Matte::Matte(const Vector3 &_color) : color(_color) {}

__device__ bool Matte::scatter(const Ray& r_in, const HitRecord& rec, Vector3& attenuation, Ray& scattered, hiprandState* local_random_state) const {
    auto dir = Vector3::random_unit_vector(local_random_state) + rec.normal;
    scattered = Ray(rec.point, dir);
    attenuation = this->color;
    return true;
}

__global__ void create_matte(Vector3 color, Material **list, size_t index) {
    list[index] = new Matte(color);
}